﻿
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

__global__ void addKernel(float*c, const float *a, const float *b)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;	
    c[threadId] = a[threadId] + b[threadId];
}

int main()
{
    float *d_a, *d_b, *d_c;
    int N = 512;
    int threadsPerBlock = 256;
    int blocks = N / threadsPerBlock;
    float* a = (float*)malloc(sizeof(float) * N);
    float* b = (float*)malloc(sizeof(float) * N);
    float* c = (float*)malloc(sizeof(float) * N);   

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_c, sizeof(float) * N);
    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);



    // Add vectors in parallel.
    addKernel <<< blocks, threadsPerBlock >>> (d_c, d_a, d_b);
    hipDeviceSynchronize();
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    hipMemcpy(c, d_c, sizeof(float) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
    {
        printf("%f ,", c[i]);
    }

    hipFree(d_c);
    hipFree(d_a);
    hipFree(d_b);
    free(a);
    free(b);
    free(c);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

